#include "hip/hip_runtime.h"
/*
 *    This file is part of CNewt.
 *
 *    CNewt is free software: you can redistribute it and/or modify
 *    it under the terms of the GNU General Public License as published by
 *    the Free Software Foundation, either version 3 of the License, or
 *    (at your option) any later version.
 *
 *    CNewt is distributed in the hope that it will be useful,
 *    but WITHOUT ANY WARRANTY; without even the implied warranty of
 *    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *    GNU General Public License for more details.
 *
 *    You should have received a copy of the GNU General Public License
 *    along with CNewt.  If not, see <http://www.gnu.org/licenses/>.
 */
extern "C" {
#include "common.h"
}

void multi_eval_sequential(uint m, uint n, void (*f)(float *, float *),
                           uint l, float *X, float *Y)
{
	for (uint j = 1; j <= l; j++) {
		f(M_COL(X, m + 1, j), M_COL(Y, n, j));
	}
}

__global__ void eval_fct_kernel(const float *X, float *Y,
                                uint m, uint n, uint l)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i <= l) {
		const float *in = &X[(m + 1) * i];
		float *out = &Y[n * i];
		float x1 = V_IDX(in, 1);
		float x2 = V_IDX(in, 2);
		V_IDX(out, 1) = (x1 * x1 + x2 * x2);
		V_IDX(out, 1) += sin(10000.f * x1) * sin(10000.f * x2) / 100.f;
	}
}

void multi_eval_gpu(uint m, uint n, uint l, float *X, float *Y)
{
	// Load X to device memory
	uint sizeX = (m + 1) * l * sizeof(float);
	float *devX = NULL;
	hipMalloc(&devX, sizeX);	
	hipMemcpy(devX, X, sizeX, hipMemcpyHostToDevice);

	// Allocate Y in device memory
	uint sizeY = n * l * sizeof(float);
	float *devY = NULL;
	hipMalloc(&devY, sizeY);

	// Invoke kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (l + threadsPerBlock - 1) / threadsPerBlock;
	eval_fct_kernel<<<blocksPerGrid, threadsPerBlock>>>(devX, devY, m, n, l);

	// Read Y from device memory
	hipMemcpy(Y, devY, sizeY, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(devX);
	hipFree(devY);
}

/**
 * multi_eval() - evaluates a function at multiple points
 * @m:              Number of parameters of the function.
 * @f:              The function to evaluate.
 * @l:              Number of points.
 * @X:              Coordinates of the points, one point per column.
 * @y:              Vector in which to store the result.
 *
 * This function assumes COLUMN-MAJOR ORDER.
 */
extern "C" void multi_eval(uint m, uint n, void (*f)(float *, float *),
                           uint l, float *X, float *Y)
{
	//multi_eval_sequential(m, n, f, l, X, Y);
	multi_eval_gpu(m, n, l, X, Y);
}
